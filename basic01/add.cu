
#include <hip/hip_runtime.h>
#include <stdio.h>
void __global__ add ( double *d_x,  double *d_y,  double *d_z);
double __device__ add_device(const double x, double y); //device修饰可以有返回值
double a = 1.3;
double b = 1.4;
double c = 2.7;
//global是host调用，可以看作是device的入口。
int main(){

    int N = 10000000;
    double size = sizeof(double)*N;
    double *h_x = (double *)malloc(size);
    double *h_y = (double *)malloc(size);
    double *h_z = (double *)malloc(size);



    for (int i=0;i<N;i++){
        h_x[i] = a;
        h_y[i] = b;
    }
    double *d_x, *d_y, *d_z; 
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);
    hipMalloc((void **)&d_z, size);

    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice); 
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);
    int block_size = 128;
    int grid_size = (N-1)/block_size + 1; // 为了当除不整的时候，多分配一个块，比如11个数，每个块大小5，就分三个
    add<<<grid_size, block_size>>>(d_x, d_y, d_z);//也可以直接传入N或者size
    hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost);
  //  check(h_z, N);
    double sum = 0.0;
    for (int i=0;i<N;i++){
        sum += h_z[i];
    }
    hipDeviceSynchronize();
    printf("the sum is %f,%f", sum,h_z[0]);
    free(h_x);
    free(h_y);
    free(h_z);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    return 0;
}

void __global__ add( double *d_x,  double *d_y,  double *d_z){

    const int n = blockDim.x*blockIdx.x+threadIdx.x;
    d_z[n] = add_device(d_x[n] , d_y[n]);
}
double __device__ add_device(const double x, const double y){
    return x+y;
}