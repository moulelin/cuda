#include "hip/hip_runtime.h"
#include <stdio.h>
void __global__ add (const double *d_x, const double *d_y, const double *d_z);
void __device__ check(const double x, double y);
int a = 1.3;
int b = 1.4;
int c = 2.7;
//global是host调用，可以看作是device的入口。
int main(){

    int N = 1000000;
    double size = sizeof(double)*N;
    double *h_x = (double *)malloc(size);
    double *h_y = (double *)malloc(size);
    double *h_z = (double *)malloc(size);



    for (int i=0;i<N;i++){
        h_x[i] = a;
        h_y[i] = b;
    }
    double *d_x, *d_y, *d_z; 
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);
    hipMalloc((void **)&d_z, size);

    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);
    block_size = 128;
    grid_size = (N-1)/block_size +1;
    add<<<grid_size, block_size>>>(d_x, d_y, d_z);
    hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost);
    check(h_z, N);
    double sum = 0;
    for (int i=0;i<N;i++){
        sum += h_z[i];
    }
    printf("the sum is %f", sum);
    return 0;
}

void __global__ add(const double *d_x, const double *d_y, const double *d_z){

    const int n = blockDim.x*blockIdx.x+threadIdx.x;
    d_z[n] = add_device(d_x[n] + d_y[n]);
}
double __device__ add_device(const double x, const double y){
    return x+y;
}