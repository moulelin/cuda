
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void hello_from_gpu(){
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    printf("this is from gpu and the block is %d, the thread is %d\n", bid, tid);
}
int main(void){
    hello_from_gpu<<<2,3>>>();
    hipDeviceSynchronize();
    return 0;
}